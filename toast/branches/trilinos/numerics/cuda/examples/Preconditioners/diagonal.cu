#include <cusp/precond/diagonal.h>
#include <cusp/krylov/bicgstab.h>
#include <cusp/krylov/cg.h>
#include <cusp/csr_matrix.h>
#include <cusp/io/matrix_market.h>

#include <iostream>

// where to perform the computation
typedef cusp::device_memory MemorySpace;

// which floating point type to use
typedef float ValueType;

int main(void)
{
    // create an empty sparse matrix structure (HYB format)
    cusp::csr_matrix<int, ValueType, MemorySpace> A;

    // load a matrix stored in MatrixMarket format
    cusp::io::read_matrix_market_file(A, "A.mtx");

    // Note: A has poorly scaled rows & columns

    // solve without preconditioning
    {
        std::cout << "\nSolving with no preconditioner" << std::endl;
    
        // allocate storage for solution (x) and right hand side (b)
        cusp::array1d<ValueType, MemorySpace> x(A.num_rows, 0);
        cusp::array1d<ValueType, MemorySpace> b(A.num_rows, 1);

        // set stopping criteria (iteration_limit = 100, relative_tolerance = 1e-6)
        cusp::verbose_monitor<ValueType> monitor(b, 100, 1e-6);
        
        // solve
        cusp::krylov::cg(A, x, b, monitor);
    }

    // solve with diagonal preconditioner
    {
        std::cout << "\nSolving with diagonal preconditioner (M = D^-1)" << std::endl;
        
        // allocate storage for solution (x) and right hand side (b)
        cusp::array1d<ValueType, MemorySpace> x(A.num_rows, 0);
        cusp::array1d<ValueType, MemorySpace> b(A.num_rows, 1);

        // set stopping criteria (iteration_limit = 100, relative_tolerance = 1e-6)
        cusp::verbose_monitor<ValueType> monitor(b, 100, 1e-6);

        // setup preconditioner
        cusp::precond::diagonal<ValueType, MemorySpace> M(A);

        // solve
        cusp::krylov::cg(A, x, b, monitor, M);
    }

    return 0;
}

